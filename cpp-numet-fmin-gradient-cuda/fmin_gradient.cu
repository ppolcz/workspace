#include "hip/hip_runtime.h"
/* 
 * File:   main.cu
 * Author: Péter Polcz <ppolcz@gmail.com>
 *
 * Created on February 13, 2015, 10:15 AM
 */

// System includes
#include <iostream>
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_string.h>
#include <helper_functions.h>

#ifndef __CUDA_ENABLED__
#define blockIdx
#define blockDim
#define threadIdx
#define atomicExch
#define atomicCAS
#define atomicAdd
#endif

//__device__ __forceinline__ float func(float x, float y)
//{
//    return x * x + y * y; // 18 clock cycles ( - clock()'s clock cycles )
//}
//
//__device__ __forceinline__ void dfunc(float x, float y, float &dx, float &dy)
//{
//    dx = 2*x;
//    dy = 2*y;
//}

/*
 * Matlab: 
[x,y] = meshgrid([-10:0.1:10]); surf(x, y, sin(3*x) + sin(y)), view(0,90), shading interp, hold on, plot3(-0.523599, -1.482375, 100, 'sr') 
 */
//__device__ __forceinline__ float func(float x, float y)
//{
//    return sin(3*x) + sin(y); // ~764 clock cycles
//}
//
//__device__ __forceinline__ void dfunc(float x, float y, float &dx, float &dy)
//{
//    dx = 3*cos(3*x);
//    dy = cos(y);
//}

/*
 * Matlab: 
[x,y] = meshgrid([-3:0.1:3]); surf(x, y, sin(2*x) + sin(3*y) + (x+1).^2 + y), shading interp, hold on, plot3(-0.857750, -0.636904, 100, 'sr'), plot3(-0.857553, -2.731273, 100, 'sr'), view(0,90)
 */
__device__ __forceinline__ float func(float x, float y)
{
    return sin(2*x) + sin(3*x) + (x+1)*(x+1) + y; // ~700 clock cycles - trigonometric functions
}

__device__ __forceinline__ void dfunc(float x, float y, float &dx, float &dy)
{
    dx = 2*cos(2*x) + 2*(x+1);
    dy = 3*cos(3*y) + 1;
}

/* multiply gradient with gamma scalar */
__device__ __forceinline__ 
void dfunc_gamma(float x, float y, float &dx, float &dy, float gamma)
{
    dfunc(x,y,dx,dy);
    dx *= gamma;
    dy *= gamma;
}

/* meaning the will produce some 128 bit load instead of several 32 bit load */
__device__ __align__(16) float variable[1024];

__device__ __align__(16) float4 result;
__device__ int lock;

__global__ void gradient_search(
    float *_x, float *_y,
    int size, float gamma, float precision, int iternum)
{
    atomicExch(&lock, 0);
    
    // arguments: 6x 32bit (the pointers could be 64bit)
    
    //float4 xx = make_float4(1,2,3,4);
    //float4 yy = make_float4(3,4,5,6);
    //float4 zz = xx + yy; // not oke
    
    float x, y, dx, dy, x_old, y_old, tmp; 
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        x = _x[i];
        y = _y[i];
    }
    
    x_old = x;
    y_old = y;
    
    /* gradient descent iteration */
    int it = 0;
    float diff = precision + 1;
    for (; it < iternum && precision < diff; ++it)
    {
        // gradient search step
        dfunc_gamma(x, y, dx, dy, gamma);
        x -= dx;
        y -= dy;

        // calculate Euclidean distance between [x, y] and [x_old, y_old]
        diff = x_old - x;
        diff *= diff;
        tmp = y_old - y;
        tmp *= tmp;
        diff += tmp;
        diff = sqrt(diff);
        
        // store new function arguments
        x_old = x;
        y_old = y;
    }
    
    int k = 0;
    float fmin = func(x,y);

    /* if it == iternum, the gradient method has not found a good solution */
    if (it < iternum)
    {
        /* if the gradient search succeeded, then update the result, 
         * thread safety is critical */
        
        // IMPORTANT: this is a good and safe solution for mutual exclusion
        bool wait = true;
        int max = 10000;
        for (; k < max && wait; ++k) {
            if (atomicCAS(&lock, 0, 1) == 0) {

                // thread safe code [BEGIN]

                if (result.z > fmin) 
                {
                    result.x = x;
                    result.y = y;
                    result.z = fmin;
                }

                // thread safe code [END]
                
                wait = false;
                atomicExch(&lock, 0);
            }
        } 
    }
    

    // IMPORTANT: this is not a working solution for mutual exclusion, causes hard freezing
    // --
    // for (int k = 0; atomicCAS(&lock, 0, 1) && k < 100; ++k);
    // 
    // printf("thread %d: %d\n", threadIdx.x, result.x);
    // result.x += 1;
    // 
    // atomicExch(&lock,0);

    printf("x = [%f, %f], f = %f, iteration = %d, k = %d\n", x, y, fmin, it-1, k);
    
    if (blockIdx.x * blockDim.x + threadIdx.x == 0)
    {
        printf("\nbest: x = [%f, %f], f = %f\n\n", result.x, result.y, result.z);
    }
}

__global__ void gradient_search_dt(
    float *_x, float *_y, float *_r, int4 *_t,
    int dtnr, int size, float gamma, float precision, int iternum)
{
    clock_t t0 = clock(), t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12, t13, t14, t15, t16;
    
    atomicExch(&lock, 0);
    t1 = clock(); // 22
    
    float x, y, dx, dy, x_old, y_old, tmp; 
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    x = _x[i];
    y = _y[i];
    t2 = clock(); // 500-700 - global memory access
    
    x_old = x;
    y_old = y;
    t3 = clock(); // 7 - this takes effectively no time
    
    /* gradient descent iteration */
    int it = 0;
    float diff = precision;
    diff += 123;
    t4 = clock(); // 6
    
    diff += 0.1;
    t5 = clock(); // 146-147 - floating point addition !!!
    
    for (; it < iternum && precision < diff; ++it)
    {
        // gradient search step
        dfunc_gamma(x, y, dx, dy, gamma);
        x -= dx;
        y -= dy;

        // calculate Euclidean distance between [x, y] and [x_old, y_old]
        diff = x_old - x;
        diff *= diff;
        tmp = y_old - y;
        tmp *= tmp;
        diff += tmp;
        diff = sqrt(diff);
        
        // store new function arguments
        x_old = x;
        y_old = y;
    }
    t6 = clock(); // 7000-9000
    
    int k = 0;
    float fmin = func(x,y);
    t7 = clock(); // it depends on func(...)

    /* if it == iternum, the gradient method has not found a good solution */
    if (it < iternum)
    {
        /* if the gradient search succeeded, then update the result, 
         * thread safety is critical */
        
        // IMPORTANT: this is a good and safe solution for mutual exclusion
        bool wait = true;
        int max = 10000;
        t9 = clock();
        for (; k < max && wait; ++k) {
            t10 = clock();
            if (atomicCAS(&lock, 0, 1) == 0) {

                // thread safe code [BEGIN]
                if (_r[3] > fmin) 
                {
                    _r[0] = x;
                    _r[1] = y;
                    _r[3] = fmin;
                }

                // thread safe code [END]
                
                wait = false;
                atomicExch(&lock, 0);
            }
            t11 = clock(); // t11 - t10 ~ equal on all threads
        } 
    }
    t8 = clock();

    t12 = clock();
    if (i == 0)
    {
        x = 0;
        x += sin(sqrt(0.123));
        x -= 0.1;
        x += -0.023;
    } 
    else if (i == 1)
    {
        x = 0;
        x += sqrt(sin(0.243));
    }
    else if (i == 2)
    {
        x = 0;
        x += sqrt(sin(0.243));
    }
    else if (i == 3)
    {
        x = 0;
        x += sqrt(sin(0.243));
    }
    t13 = clock();
    
    _t[dtnr * i + 0] = make_int4(t1 - t0, t2 - t1, t3 - t2, t4 - t3);
    _t[dtnr * i + 1] = make_int4(t5 - t4, t6 - t5, t7 - t6, t8 - t7);
    _t[dtnr * i + 2] = make_int4(t11 - t9, t11 - t10, 0, t8 - t0);
    _t[dtnr * i + 3] = make_int4(t13 - t12, float_as_int(x), it-1, k-1);
}

__global__ void gradient_search(
    float *_x, float *_y, float *_r, int4 *_t,
    int size, float gamma, float precision, int iternum)
{
    clock_t t0 = clock(), t1, t2, t3, t4, t5, t6, t7, t8;
    
    atomicExch(&lock, 0);
    t1 = clock();
    
    float x, y, dx, dy, x_old, y_old, tmp; 
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    x = _x[i];
    y = _y[i];
    t2 = clock();
    
    x_old = x;
    y_old = y;
    t3 = clock();
    
    /* gradient descent iteration */
    int it = 0;
    float diff = precision + 1;
    for (; it < iternum && precision < diff; ++it)
    {
        // gradient search step
        dfunc_gamma(x, y, dx, dy, gamma);
        x -= dx;
        y -= dy;

        // calculate Euclidean distance between [x, y] and [x_old, y_old]
        diff = x_old - x;
        diff *= diff;
        tmp = y_old - y;
        tmp *= tmp;
        diff += tmp;
        diff = sqrt(diff);
        
        // store new function arguments
        x_old = x;
        y_old = y;
    }
    t4 = clock();
    
    int k = 0;
    float fmin = func(x,y);

    /* if it == iternum, the gradient method has not found a good solution */
    if (it < iternum)
    {
        /* if the gradient search succeeded, then update the result, 
         * thread safety is critical */
        
        // IMPORTANT: this is a good and safe solution for mutual exclusion
        bool wait = true;
        int max = 10000;
        t5 = clock();
        for (; k < max && wait; ++k) {
            if (atomicCAS(&lock, 0, 1) == 0) {

                // thread safe code [BEGIN]
                t6 = clock();
                if (_r[3] > fmin) 
                {
                    _r[0] = x;
                    _r[1] = y;
                    _r[3] = fmin;
                }

                // thread safe code [END]
                
                wait = false;
                atomicExch(&lock, 0);
            }
        } 
    }
    t7 = clock();
    t8 = clock();

    _t[2 * i + 0] = make_int4(t1 - t0, t2 - t0, t3 - t0, t4 - t0);
    _t[2 * i + 1] = make_int4(t5 - t0, t6 - t0, t7 - t0, t8 - t0);
}

template<int N>
int array_size(float (&array)[N])
{
    return N;
}

void gradient_search_launch()
{
    float x[] = { -1, 1, 1, -1, 2, 2, -2, -2, -2, -2, 2, 2 };
    float y[] = { -1, -1, 1, 1, 2, 1, 2, 1, -2, -1, -2, -2 };
    printf("array size = %d\n", array_size(x));
    
    float *d_x, *d_y, *d_r;
    int4 *d_dt;
    
    const int size = array_size(x);
    const int dtnr = 16;
    int mem_size = sizeof(float) * size;
    int dt_size = sizeof(int) * dtnr * size;

    int dt[size * dtnr];
    
    hipMalloc((void**) &d_x, mem_size);
    hipMalloc((void**) &d_y, mem_size);
    hipMalloc((void**) &d_r, 16);
    hipMalloc((void**) &d_dt, dt_size);
    
    hipMemcpy(d_x, x, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(d_dt, dt, dt_size, hipMemcpyHostToDevice);
    
    gradient_search_dt<<<1,size>>>(d_x, d_y, d_r, d_dt, dtnr/4, size, 0.1, 0.001, 1000);
    
    float res[4];
    
    hipError_t error;
    hipEvent_t stop;
    
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
    
    // Record the stop event
    error = hipEventRecord(stop, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

//    float msecTotal = 0.0f;
//    error = hipEventElapsedTime(&msecTotal, start, stop);
//
//    if (error != hipSuccess)
//    {
//        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
//        exit(EXIT_FAILURE);
//    }
//
//    // Compute and print the performance
//    float msecPerMatrixMul = msecTotal / nIter;
//    double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
//    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
//    printf(
//        "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
//        gigaFlops,
//        msecPerMatrixMul,
//        flopsPerMatrixMul,
//        threads.x * threads.y);

    // Copy result from device to host
    hipMemcpy(res, d_r, array_size(res) * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(dt, d_dt, dt_size, hipMemcpyDeviceToHost);
    
    std::cout << "result:\n";
    printf("x = [%f, %f], f = %f\n", res[0], res[1], res[3]);

    for (int i = 0; i < size; ++i)
    {
        printf("thread %d, clock cycles: [ ", i);
        for (int j = 0; j < dtnr; ++j)
        {
            if (j % 4 == 0 && j != 0) std::cout << "| ";
            std::cout << dt[i*dtnr + j] << " ";
        }
        std::cout << "]\n";
    }
    
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_r);
    hipFree(d_dt);
}

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template <int BLOCK_SIZE> 
__global__ void matrixMulCUDA(float *C, float *A, float *B, int wA, int wB)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
         a <= aEnd;
         a += aStep, b += bStep)
    {

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
#pragma unroll

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            Csub += As[ty][k] * Bs[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}

void constantInit(float *data, int size, float val)
{
    for (int i = 0; i < size; ++i)
    {
        data[i] = val;
    }
}

/**
 * Run a simple test of matrix multiplication using CUDA
 */
int matrixMultiply(int argc, char **argv, int block_size, dim3 &dimsA, dim3 &dimsB)
{
    // Allocate host memory for matrices A and B
    unsigned int size_A = dimsA.x * dimsA.y;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float *h_A = (float *)malloc(mem_size_A);
    unsigned int size_B = dimsB.x * dimsB.y;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float *h_B = (float *)malloc(mem_size_B);

    // Initialize host memory
    const float valB = 0.01f;
    constantInit(h_A, size_A, 1.0f);
    constantInit(h_B, size_B, valB);

    // Allocate device memory
    float *d_A, *d_B, *d_C;

    // Allocate host matrix C
    dim3 dimsC(dimsB.x, dimsA.y, 1);
    unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
    float *h_C = (float *) malloc(mem_size_C);

    if (h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host matrix C!\n");
        exit(EXIT_FAILURE);
    }

    hipError_t error;

    error = hipMalloc((void **) &d_A, mem_size_A);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_A returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_B, mem_size_B);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_B returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMalloc((void **) &d_C, mem_size_C);

    if (error != hipSuccess)
    {
        printf("hipMalloc d_C returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // copy host memory to device
    error = hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_A,h_A) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (d_B,h_B) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // Setup execution parameters
    dim3 threads(block_size, block_size);
    dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

    // Create and start timer
    printf("Computing result using CUDA Kernel...\n");

    // Performs warmup operation using matrixMul CUDA kernel
    if (block_size == 16)
    {
        matrixMulCUDA<16><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }
    else
    {
        matrixMulCUDA<32><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
    }

    printf("done\n");

    hipDeviceSynchronize();

    // Allocate CUDA events that we'll use for timing
    hipEvent_t start;
    error = hipEventCreate(&start);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    hipEvent_t stop;
    error = hipEventCreate(&stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Record the start event
    error = hipEventRecord(start, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Execute the kernel
    int nIter = 300;

    for (int j = 0; j < nIter; j++)
    {
        if (block_size == 16)
        {
            matrixMulCUDA<16><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
        else
        {
            matrixMulCUDA<32><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
        }
    }

    // Record the stop event
    error = hipEventRecord(stop, NULL);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Wait for the stop event to complete
    error = hipEventSynchronize(stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    float msecTotal = 0.0f;
    error = hipEventElapsedTime(&msecTotal, start, stop);

    if (error != hipSuccess)
    {
        fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Compute and print the performance
    float msecPerMatrixMul = msecTotal / nIter;
    double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
    printf(
        "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
        gigaFlops,
        msecPerMatrixMul,
        flopsPerMatrixMul,
        threads.x * threads.y);

    // Copy result from device to host
    error = hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (h_C,d_C) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    printf("Checking computed result for correctness: ");
    bool correct = true;

    // test relative error by the formula
    //     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
    double eps = 1.e-6 ; // machine zero

    for (int i = 0; i < (int)(dimsC.x * dimsC.y); i++)
    {
        double abs_err = fabs(h_C[i] - (dimsA.x * valB));
        double dot_length = dimsA.x;
        double abs_val = fabs(h_C[i]);
        double rel_err = abs_err/abs_val/dot_length ;

        if (rel_err > eps)
        {
            printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i, h_C[i], dimsA.x*valB, eps);
            correct = false;
        }
    }

    printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

    // Clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    printf("\nNote: For peak performance, please refer to the matrixMulCUBLAS example.\n");

    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset();

    if (correct)
    {
        return EXIT_SUCCESS;
    }
    else
    {
        return EXIT_FAILURE;
    }
}


/**
 * Program main
 */
int main_gradient(int argc, char** argv)
{
    printf("[Matrix Multiply Using CUDA] - Starting...\n");

    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
        checkCmdLineFlag(argc, (const char **)argv, "?"))
    {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
        printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
        printf("  Note: Outer matrix dimensions of A & B matrices must be equal.\n");

        exit(EXIT_SUCCESS);
    }

    // By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
    int devID = 0;

    if (checkCmdLineFlag(argc, (const char **)argv, "device"))
    {
        devID = getCmdLineArgumentInt(argc, (const char **)argv, "device");
        hipSetDevice(devID);
    }

    hipError_t error;
    hipDeviceProp_t deviceProp;
    error = hipGetDevice(&devID);

    if (error != hipSuccess)
    {
        printf("hipGetDevice returned error code %d, line(%d)\n", error, __LINE__);
    }

    error = hipGetDeviceProperties(&deviceProp, devID);

    if (deviceProp.computeMode == hipComputeModeProhibited)
    {
        fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
        exit(EXIT_SUCCESS);
    }

    if (error != hipSuccess)
    {
        printf("hipGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
    }
    else
    {
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }

    // Use a larger block size for Fermi and above
    int block_size = (deviceProp.major < 2) ? 16 : 32;

    dim3 dimsA(5*2*block_size, 5*2*block_size, 1);
    dim3 dimsB(5*4*block_size, 5*2*block_size, 1);

    // width of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "wA"))
    {
        dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
    }

    // height of Matrix A
    if (checkCmdLineFlag(argc, (const char **)argv, "hA"))
    {
        dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
    }

    // width of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "wB"))
    {
        dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
    }

    // height of Matrix B
    if (checkCmdLineFlag(argc, (const char **)argv, "hB"))
    {
        dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
    }

    if (dimsA.x != dimsB.y)
    {
        printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
               dimsA.x, dimsB.y);
        exit(EXIT_FAILURE);
    }

    printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x, dimsB.y);

    gradient_search_launch();
    int matrix_result = matrixMultiply(argc, argv, block_size, dimsA, dimsB);

    exit(matrix_result);
}

